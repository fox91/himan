#include "cuda_plugin_helper.h"
#include "plugin_factory.h"
#include <NFmiGribPacking.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/system/cuda/execution_policy.h>

#define HIMAN_AUXILIARY_INCLUDE

#include "cache.h"
#include "fetcher.h"

#undef HIMAN_AUXILIARY_INCLUDE

namespace himan
{
namespace cuda
{
bool Unpack(std::shared_ptr<himan::info<double>> fullInfo, hipStream_t& stream, double* d_arr)
{
	return Unpack<double>(fullInfo, stream, d_arr);
}

template <typename T>
bool Unpack(std::shared_ptr<himan::info<T>> fullInfo, hipStream_t& stream, T* d_arr)
{
	using namespace himan;
	using namespace himan::plugin;

	const size_t N = fullInfo->SizeLocations();

	ASSERT(N > 0);
	ASSERT(d_arr);

	// Unpack if needed, leave data to device
	if (fullInfo->PackedData()->HasData())
	{
		ASSERT(fullInfo->PackedData()->packingType == kSimplePacking);
		ASSERT(fullInfo->Data().Size() == N);

		const auto pck = std::dynamic_pointer_cast<simple_packed>(fullInfo->PackedData());

		return NFmiGribPacking::simple_packing::Unpack<T>(d_arr, pck->data, pck->bitmap, pck->unpackedLength,
		                                               pck->packedLength, pck->coefficients, stream);
		return true;
	}
	else
	{
		// Data was not packed, ie it was returned to us from cache
		CUDA_CHECK(
		    hipMemcpyAsync(d_arr, fullInfo->Data().ValuesAsPOD(), sizeof(T) * N, hipMemcpyHostToDevice, stream));

		CUDA_CHECK(hipStreamSynchronize(stream));
		return false;
	}
}

template bool Unpack<double>(std::shared_ptr<himan::info<double>>, hipStream_t&, double*);
template bool Unpack<float>(std::shared_ptr<himan::info<float>>, hipStream_t&, float*);

template <typename T>
void PrepareInfo(std::shared_ptr<himan::info<T>> info, T* d_ret, hipStream_t& stream, bool copyToHost)
{
	if (Unpack(info, stream, d_ret) && copyToHost)
	{
		CUDA_CHECK(hipMemcpyAsync(info->Data().ValuesAsPOD(), d_ret, sizeof(T) * info->SizeLocations(),
		                           hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipStreamSynchronize(stream));

		info->PackedData()->Clear();

		auto c = GET_PLUGIN(cache);
		c->Insert(info);
	}
}

template void PrepareInfo<double>(std::shared_ptr<himan::info<double>>, double*, hipStream_t&, bool);
template void PrepareInfo<float>(std::shared_ptr<himan::info<float>>, float*, hipStream_t&, bool);

template <typename T>
void ReleaseInfo(std::shared_ptr<himan::info<T>> info, T* d_arr, hipStream_t& stream)
{
	CUDA_CHECK(hipMemcpyAsync(info->Data().ValuesAsPOD(), d_arr, info->SizeLocations() * sizeof(T),
	                           hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));
}

template void ReleaseInfo<double>(std::shared_ptr<himan::info<double>>, double*, hipStream_t&);
template void ReleaseInfo<float>(std::shared_ptr<himan::info<float>>, float*, hipStream_t&);

template <typename T>
std::shared_ptr<himan::info<T>> Fetch(const std::shared_ptr<const plugin_configuration> conf,
                                      const himan::forecast_time& theTime, const himan::level& theLevel,
                                      const himan::params& theParams, const himan::forecast_type& theType,
                                      bool returnPacked)
{
	for (const auto& p : theParams)
	{
		auto ret = Fetch<T>(conf, theTime, theLevel, p, theType, returnPacked);

		if (ret)
		{
			return ret;
		}
	}
	return nullptr;
}

template std::shared_ptr<himan::info<double>> Fetch<double>(const std::shared_ptr<const plugin_configuration>,
                                                            const himan::forecast_time&, const himan::level&,
                                                            const himan::params&, const himan::forecast_type&, bool);
template std::shared_ptr<himan::info<float>> Fetch<float>(const std::shared_ptr<const plugin_configuration>,
                                                          const himan::forecast_time&, const himan::level&,
                                                          const himan::params&, const himan::forecast_type&, bool);

template <typename T>
std::shared_ptr<himan::info<T>> Fetch(const std::shared_ptr<const plugin_configuration> conf,
                                      const himan::forecast_time& theTime, const himan::level& theLevel,
                                      const himan::param& theParam, const himan::forecast_type& theType,
                                      bool returnPacked)
{
	try
	{
		auto f = GET_PLUGIN(fetcher);
		return f->Fetch<T>(conf, theTime, theLevel, theParam, theType, returnPacked);
	}
	catch (HPExceptionType& e)
	{
		if (e != kFileDataNotFound)
		{
			throw std::runtime_error("cape_cuda::Fetch(): Unable to proceed");
		}

		return nullptr;
	}
}

template std::shared_ptr<himan::info<double>> Fetch<double>(const std::shared_ptr<const plugin_configuration>,
                                                            const himan::forecast_time&, const himan::level&,
                                                            const himan::param&, const himan::forecast_type&, bool);
template std::shared_ptr<himan::info<float>> Fetch<float>(const std::shared_ptr<const plugin_configuration>,
                                                          const himan::forecast_time&, const himan::level&,
                                                          const himan::param&, const himan::forecast_type&, bool);

}  // namespace cuda
}  // namespace himan
